#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 16
#define TILE_SIZE 16

__global__ void matrixMult(float *A, float *B, float *C, int size);

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Usage: %s size\n", argv[0]);
        exit(1);
    }

    int size = atoi(argv[1]);
    float input_size = size * size * sizeof(float);
    if (size <= 0)
    {
        printf("Invalid matrix size: %d\n", size);
        exit(1);
    }

    // Allocate memory for matrices A, B, and C on the host
    float *A = (float *)malloc(input_size);
    float *B = (float *)malloc(input_size);
    float *C = (float *)malloc(input_size);

    // Allocate memory for matrices A, B, and C on the device
    float *dev_a, *dev_b, *dev_c;
    hipMalloc(&dev_a, input_size);
    hipMalloc(&dev_b, input_size);
    hipMalloc(&dev_c, input_size);

    // Load matrices A and B with random numbers
    srand(42);
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            A[i * size + j] = (float)rand() / (float)RAND_MAX;
            B[i * size + j] = (float)rand() / (float)RAND_MAX;
        }
    }

    // Copy matrices A and B from host to device
    hipMemcpy(dev_a, A, input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, B, input_size, hipMemcpyHostToDevice);

    // Define the grid and block dimensions for the MatrixMultKernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((size + BLOCK_SIZE - 1) / BLOCK_SIZE, (size + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Create CUDA events to measure the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Call the MatrixMultKernel on the device
    hipEventRecord(start);
    matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, size);
    hipEventRecord(stop);

    // Copy matrix C from device to host
    hipMemcpy(C, dev_c, input_size, hipMemcpyDeviceToHost);

    // Calculate the elapsed time in seconds
    float elapsedTime;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print the execution time
    printf("Execution time: %f ms\n", elapsedTime);
     FILE *csv_file;
    char csv_filename[100];
    sprintf(csv_filename, "CSV/Tiled.csv");
    csv_file = fopen(csv_filename, "a");
    fprintf(csv_file, "%d,%f\n", size, elapsedTime);
    fclose(csv_file);

    // Free memory
    free(A);
    free(B);
    free(C);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

__global__ void matrixMult(float *A, float *B, float *C, int size)
{
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;

    for (int k = 0; k < size / TILE_SIZE; k++)
    {
        // Load tiles into shared memory
        As[ty][tx] = A[row * size + k * TILE_SIZE + tx];
        Bs[ty][tx] = B[(k * TILE_SIZE + ty) * size + col];

        // Synchronize to ensure all tiles are loaded
        __syncthreads();

        // Multiply the tiles and accumulate the result
        for (int i = 0; i < TILE_SIZE; i++)
        {
            sum += As[ty][i] * Bs[i][tx];
        }

        // Synchronize to ensure all tiles are used before overwriting shared memory
        __syncthreads();
    }

    // Write the result to the output matrix
    if (row < size && col < size)
    {
        C[row * size + col] = sum;
    }
}
