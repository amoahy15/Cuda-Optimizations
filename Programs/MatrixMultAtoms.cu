#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 16

__global__ void matrixMult(float *A, float *B, float *C, int size);

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Usage: %s size\n", argv[0]);
        exit(1);
    }

    int size = atoi(argv[1]);
    float input_size = size * size * sizeof(float);
    if (size <= 0)
    {
        printf("Invalid matrix size: %d\n", size);
        exit(1);
    }

    // Allocate memory for matrices A, B, and C on the host
    float *A = (float *)malloc(input_size);
    float *B = (float *)malloc(input_size);
    float *C = (float *)malloc(input_size);

    // Allocate memory for matrices A, B, and C on the device
    float *dev_a, *dev_b, *dev_c;
    hipMalloc(&dev_a, input_size);
    hipMalloc(&dev_b, input_size);
    hipMalloc(&dev_c, input_size);

    // Load matrices A and B with random numbers
    srand(42);
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            A[i * size + j] = (float)rand() / (float)RAND_MAX;
            B[i * size + j] = (float)rand() / (float)RAND_MAX;
        }
    }

    // Copy matrices A and B from host to device
    hipMemcpy(dev_a, A, input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, B, input_size, hipMemcpyHostToDevice);

    // Define the grid and block dimensions for the MatrixMultKernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((size + BLOCK_SIZE - 1) / BLOCK_SIZE, (size + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Create CUDA events to measure the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Call the MatrixMultKernel on the device
    hipEventRecord(start);
    matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, size);
    hipEventRecord(stop);

    // Copy matrix C from device to host
    hipMemcpy(C, dev_c, input_size, hipMemcpyDeviceToHost);

    // Calculate the elapsed time in seconds
    float elapsedTime;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print the execution time
    printf("Execution time: %f ms\n", elapsedTime);
     FILE *csv_file;
    char csv_filename[100];
    sprintf(csv_filename, "CSV/Atomics.csv");
    csv_file = fopen(csv_filename, "a");
    fprintf(csv_file, "%d,%f\n", size, elapsedTime);
    fclose(csv_file);

    // Free memory
    free(A);
    free(B);
    free(C);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

__global__ void matrixMult(float *A, float *B, float *C, int size)
{
    __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for (int t = 0; t < (size + BLOCK_SIZE - 1) / BLOCK_SIZE; t++)
    {
        int tiledRow = blockIdx.y * blockDim.y + threadIdx.y;
        int tiledCol = t * blockDim.x + threadIdx.x;

        if (tiledRow < size && tiledCol < size)
            tileA[threadIdx.y][threadIdx.x] = A[tiledRow * size + tiledCol];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0;

        tiledRow = t * blockDim.y + threadIdx.y;
        tiledCol = blockIdx.x * blockDim.x + threadIdx.x;

        if (tiledRow < size && tiledCol < size)
            tileB[threadIdx.y][threadIdx.x] = B[tiledRow * size + tiledCol];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; i++)
        {
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < size && col < size)
        atomicAdd(&C[row * size + col], sum);
}
